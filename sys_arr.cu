#include "hip/hip_runtime.h"
﻿
#include "parameters.h"

Cell* sys_arr_host;
__device__ Cell* sys_arr;
dim3 grid(sys_array_size, sys_array_size);

void flush_sys_arr();
void sys_arr_free();
extern __device__ char feed_data_v(int);
extern __device__ char feed_data_h(int);
extern __global__ void _collect_result(void);
extern void sys_arr_cycle();

void sys_arr_ini()
{
	hipMalloc((void**)&sys_arr_host, sizeof(Cell) * sys_array_size * sys_array_size);
	hipMemcpyToSymbol(HIP_SYMBOL(sys_arr), &sys_arr_host, sizeof(Cell*), 0, hipMemcpyHostToDevice);
	flush_sys_arr();
	printf("Systolic array successfully initialized, size %d * %d\n", sys_array_size, sys_array_size);
}

__global__ void _flush_sys_arr()
{
	int x = blockIdx.x;
	int y = blockIdx.y;
	sys_arr[x + gridDim.x * y].x = 0;
	sys_arr[x + gridDim.x * y].y = 0;
	sys_arr[x + gridDim.x * y].x_output = 0;
	sys_arr[x + gridDim.x * y].y_output = 0;
	sys_arr[x + gridDim.x * y].result = 0;
}

void flush_sys_arr()
{
	_flush_sys_arr <<<grid, 1 >>> ();
}

void sys_arr_free()
{
	hipFree(sys_arr_host);
}

__global__ void _heart_beat()
{
	int x;
	int y;

	if (blockIdx.x == 0)
		x = feed_data_h(blockIdx.y);
	else
		x = sys_arr[blockIdx.x - 1 + blockIdx.y * gridDim.x].x;

	if (blockIdx.y == 0)
		y = feed_data_v(blockIdx.x);
	else
		y = sys_arr[blockIdx.x + (blockIdx.y - 1) * gridDim.x].y;
	__syncthreads();
	sys_arr[blockIdx.x + blockIdx.y * gridDim.x].result += x * y;
	sys_arr[blockIdx.x + blockIdx.y * gridDim.x].x = x;
	sys_arr[blockIdx.x + blockIdx.y * gridDim.x].y = y;
}

__global__ void _cell_update()
{
	sys_arr[blockIdx.x + blockIdx.y * gridDim.x].x_output = sys_arr[blockIdx.x + blockIdx.y * gridDim.x].x;
	sys_arr[blockIdx.x + blockIdx.y * gridDim.x].y_output = sys_arr[blockIdx.x + blockIdx.y * gridDim.x].y;
}

__global__ void _result_shift()
{
	int result;
	if (blockIdx.y == 0)
		result = 0;
	else
		result = sys_arr[blockIdx.x + (blockIdx.y - 1) * gridDim.x].result;
	__syncthreads();
	sys_arr[blockIdx.x + threadIdx.x * gridDim.x].result = result;
}

void heart_beat()
{
	_heart_beat <<< grid, 1 >>> ();
	sys_arr_cycle();
	hipDeviceSynchronize();
}

void result_shift()
{
	_collect_result << <grid, 1 >> > ();
	_result_shift <<< grid, 1 >>> ();
	sys_arr_cycle();
	hipDeviceSynchronize();
}