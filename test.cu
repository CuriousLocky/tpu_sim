﻿#include "control.h"
#include "simulation.h"
#include "stdio.h"
#include "stdlib.h"
#include "inttypes.h"

int main()
{
	char A[] = {
		3,2,3,
		1,4,4,
		5,6,7,
		2,9,7,
		1,2,1
	};
	char B[] = {
		1,2,3,4,
		5,6,7,8,
		4,3,2,1
	};
	int32_t result[20];
	tpu_ini();
	read_host_memory(A, 5, 3);
	read_weights(B, 3, 4);
	matrix_multiply();
	activate();
	write_host_memory(result);

	printf("computation result:\n");
	for (int i = 0; i < 5; i++) {
		for (int j = 0; j < 4; j++)
			printf("%d ", result[i*4+j]);
		printf("\n");
	}
	printf("\n");

	printSimInfo();

	char X[] = {
		0,0,0,
		0,0,0,
		0,0,0
	};
	char Y[] = {
		0,0,0,
		0,0,0,
		0,0,0
	};

	read_host_memory(X, 3, 3);
	read_weights(Y, 3, 3);
	int32_t R[6];
	matrix_convolution();
	write_host_memory(R);
	for (int i = 0; i < 1; i++) {
		for (int j = 0; j < 1; j++)
			printf("%d ", R[i * 1 + j]);
		printf("\n");
	}

	printSimInfo();
}